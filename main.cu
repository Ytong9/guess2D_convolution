#include "hip/hip_runtime.h"
/*****************************
Copyright:Yes
Author:Ytong
Date:2019.4.8
Desscription:����ʵ������������������������������Ͷ��壩�����еĳ��Զ�����������
*****************************/

#include "stdafx.cuh"
#include <omp.h>   

/*************************************************
Function:gPU_gauss_1
Description:ʹ��GPU�����ά��˹����������1
Calls:NULL
Table Accessed:�����ʵı����������ǣ�������ݿ�����ĳ���
Table Updated:���޸ĵı����������ǣ�������ݿ�����ĳ���
Input:�������double����s��������Ϊ��˹�����еĲ���s
Output:void
Return:None
Others:��Ӧ����CUDA_syn_Dimen_1
*************************************************/
__global__ void gPU_gauss_1(double s)
{
	double arr = 0;
	int row = threadIdx.x / int(6 * s + 1);
	int col = threadIdx.x % int(6 * s + 1);
	//printf("blockId:%d %d blockDim:%d %d threadId:%d %d\n",blockIdx.x,blockIdx.y,blockDim.x,blockDim.y,threadIdx.x,threadIdx.y);
	//printf("%d %d\n", row, col);
	arr = exp(-(pow(double(row) - 3 * s, 2) + pow(3 * s - double(col), 2))/ 2 / s / s)/ s / pow(2 * Pi, 0.5);
	printf("%5.4f ", arr);
}

/*************************************************
Function:gPU_gauss_2
Description:ʹ��GPU�����ά��˹����������2
Calls:NULL
Table Accessed:�����ʵı����������ǣ�������ݿ�����ĳ���
Table Updated:���޸ĵı����������ǣ�������ݿ�����ĳ���
Input:�������double����s��������Ϊ��˹�����еĲ���s
Output:void
Return:None
Others:��Ӧ����CUDA_syn_Dimen_2
*************************************************/
__global__ void gPU_gauss_2(double s)
{
	double arr = 0;
	int row = threadIdx.x;
	int col = threadIdx.y;
	//printf("blockId:%d %d blockDim:%d %d threadId:%d %d\n",blockIdx.x,blockIdx.y,blockDim.x,blockDim.y,threadIdx.x,threadIdx.y);
	//printf("%d %d\n", row, col);
	//arr = exp(-(pow(double(row) - 3 * s, 2) + pow(3 * s - double(col), 2)) / 2 / s / s) / s / pow(2 * Pi, 0.5);
	//printf("%5.4f ", arr);
}

/*************************************************
Function:gPU_gauss_3
Description:ʹ��GPU�����ά��˹����������1
Calls:NULL
Table Accessed:�����ʵı����������ǣ�������ݿ�����ĳ���
Table Updated:���޸ĵı����������ǣ�������ݿ�����ĳ���
Input:�������double����s��������Ϊ��˹�����еĲ���s
Output:void
Return:None
Others:��Ӧ����CUDA_syn_Dimen1_2
*************************************************/
__global__ void gPU_gauss_3(double s)
{
	double arr = 0;
	int row = threadIdx.x;
	int col = 0;
	//printf("blockId:%d %d blockDim:%d %d threadId:%d %d\n",blockIdx.x,blockIdx.y,blockDim.x,blockDim.y,threadIdx.x,threadIdx.y);
	//printf("%d %d\n", row, col);
		for (; col<int(6 * s + 1); col++)
		{
			arr = exp(-(pow(double(row) - 3 * s, 2) + pow(3 * s - double(col), 2)) / 2 / s / s) / s / pow(2 * Pi, 0.5);
			printf("%5.4f ", arr);
		}
}

/*************************************************
Function:cPU_gauss
Description:ʹ��CPU�����ά��˹����������1
Calls:NULL
Table Accessed:�����ʵı����������ǣ�������ݿ�����ĳ���
Table Updated:���޸ĵı����������ǣ�������ݿ�����ĳ���
Input:�������double����s��������Ϊ��˹�����еĲ���s
Output:void
Return:None
Others:2��ѭ�������м��㣨2s+1)^2��
*************************************************/
void cPU_gauss(double s)
{
	int size_x = 6 * int(s) + 1;
	int size_y = size_x;
	double arr[100][100] = {};
	//#pragma omp parallel    //�����Ƿ�ʹ��OpenMP������
	for (int i = 0; i < size_x; i++)
	{
		for (int j = 0; j < size_y; j++)
		{
			arr[i][j] = exp(-(pow(double(i) - 3 * s, 2) + pow(3 * s - double(j), 2)) / 2 / s / s) / s / pow(2 * Pi, 0.5);
			printf("%5.4f ", arr[i][j]);
		}
		printf("\n");
	}
}


/*************************************************
Function:CUDA_syn_Dimen_1
Description:����kernel��ִ�����ã�grid��block�Ĺ�񣬼�1x1x1����169x1x1����s = 2Ϊ����ÿ���̼߳���13��
Calls:gPU_gauss_1
Table Accessed:�����ʵı����������ǣ�������ݿ�����ĳ���
Table Updated:���޸ĵı����������ǣ�������ݿ�����ĳ���
Input:�������double s�����ڴ��ݸ�gPU_gauss��Ϊ����
Output:�����������˵����
Return:��������ֵ��˵��
Others:����˵��
*************************************************/
void CUDA_syn_Dimen_1(double s)
{
	dim3 grid_size(1, 1, 1);
	dim3 block_size(pow(6*s+1,2), 1, 1);
	gPU_gauss_1<<<grid_size, block_size>>>(s);
	hipDeviceSynchronize();
}

/*************************************************
Function:CUDA_syn_Dimen_2
Description:����kernel��ִ�����ã�grid��block�Ĺ�񣬼�1x1x1����13x13x1����s = 2Ϊ����ÿ���̼߳���1��
Calls:gPU_gauss_2
Table Accessed:�����ʵı����������ǣ�������ݿ�����ĳ���
Table Updated:���޸ĵı����������ǣ�������ݿ�����ĳ���
Input:�������double s�����ڴ��ݸ�gPU_gauss��Ϊ����
Output:�����������˵����
Return:��������ֵ��˵��
Others:����˵��
*************************************************/
void CUDA_syn_Dimen_2(double s)
{
	dim3 grid_size(1, 1, 1);
	dim3 block_size(6 * s + 1, 6 * s + 1, 1);
	gPU_gauss_2<<<grid_size, block_size >>>(s);
	hipDeviceSynchronize();
}


/*************************************************
Function:CUDA_syn_Dimen1_2
Description:����kernel��ִ�����ã�grid��block�Ĺ�񣬼�1x1x1����13x1x1����s = 2Ϊ����,ÿ���̼߳���13��
Calls:gPU_gauss_3
Table Accessed:�����ʵı����������ǣ�������ݿ�����ĳ���
Table Updated:���޸ĵı����������ǣ�������ݿ�����ĳ���
Input:�������double s�����ڴ��ݸ�gPU_gauss��Ϊ����
Output:�����������˵����
Return:��������ֵ��˵��
Others:����˵��
*************************************************/
void CUDA_syn_Dimen1_2(double s)
{
	dim3 grid_size(1, 1, 1);
	dim3 block_size(6 * s + 1, 1, 1);
	gPU_gauss_3 << <grid_size, block_size >> >(s);
	hipDeviceSynchronize();
}

/*************************************************
Function:main
Description:�����������ڶ���s�����ò�ͬ�ĺ����������˹��������ʱ����������
Calls:	QueryPerformanceCounter��&num��
		QueryPerformanceCounter(&num);
		cPU_gauss(s);
		CUDA_syn_Dimen_1(s);
		CUDA_syn_Dimen_2(s);
		CUDA_syn_Dimen1_2(s);
Table Accessed:�����ʵı����������ǣ�������ݿ�����ĳ���
Table Updated:���޸ĵı����������ǣ�������ݿ�����ĳ���
Input:	none
Output:�����������˵����
Return:��������ֵ��˵��
Others:����˵��
*************************************************/
int main()
{
	printf("Input the Number s:");
	double s = 0.0;
	scanf("%lf", &s);

	LARGE_INTEGER  num;
	long long int start, end, freq;
	QueryPerformanceFrequency(&num);
	freq = num.QuadPart;
	QueryPerformanceCounter(&num);
	start = num.QuadPart;
	//cPU_gauss(s);
	//CUDA_syn_Dimen_1(s);
	//CUDA_syn_Dimen_2(s);
	CUDA_syn_Dimen1_2(s);
	QueryPerformanceCounter(&num);
	end = num.QuadPart;
	printf("��˺������֣����к�ʱ��%dms\n", (end - start) * 1000 / freq);

	return 0;

}